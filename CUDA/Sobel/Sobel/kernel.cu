#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

// image dimensions WIDTH & HEIGHT
#define WIDTH 256
#define HEIGHT 256

// Block width WIDTH & HEIGHT
#define BLOCK_W 16
#define BLOCK_H 16

// buffer to read image into
float image[HEIGHT][WIDTH];

// buffer for resulting image
float final[HEIGHT][WIDTH];

// prototype declarations
void load_image();
void call_kernel();
void save_image();

// Times used to calculate performance
double start_time_blur, end_time_blur;
double start_time_sobel, end_time_sobel;
double start_time_total, end_time_total;

// pgm file reader header file
extern "C" {
#include "pgmio.h"
}

// time header file
#include "win-gettimeofday.h"

// Kernel for image bluring before sobel filter operation
__global__ void imageBlur(float *input, float *output, int width, int height) {
	
	// index of thread
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	// width of image
	int numcols = WIDTH;

	// float for pixel
	float blur;

	// if within the bounds of the image
	if (row <= height && col <= width && row > 0 && col > 0)
	{
		// weights
		int		x1,
			x3, x4, x5,
				x7;

		// blur
		// 0.0 0.2 0.0
		// 0.2 0.2 0.2
		// 0.0 0.2 0.0

		// get index of mask & pixel to edit
		x1 = input[(row + 1) * numcols + col];			// up
		x3 = input[row * numcols + (col - 1)];			// left
		x4 = input[row * numcols + col];				// center
		x5 = input[row * numcols + (col + 1)];			// right
		x7 = input[(row + -1) * numcols + col];			// down


		// calculate blur
		blur = (x1 * 0.2) + (x3 * 0.2) + (x4 * 0.2) + (x5 * 0.2) + (x7 * 0.2);

		// output pixel data
		output[row * numcols + col] = blur;
	}
}

// sobel filter
__global__ void sobelFilter(float *input, float *output, int width, int height) {
	
	// index of thread
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	/// width of image
	int numcols = WIDTH;

	float gradient_h;	// horizontal gradient
	float gradient_v;	// vertical gradient
	float gradient;		// combined gradient
	float thresh = 30;	// threshold value

	// if within the bounds of the image
	if (row <= height && col <= width && row > 0 && col > 0)
	{
		// weights
		int x0, x1, x2, 
		    x3,	    x5, 
			x6, x7, x8;

		// horizontal
		// -1  0  1
		// -2  0  2
		// -1  0  1

		// vertical 
		// -1 -2 -1
		//  0  0  0
		//  1  2  1

		// get index of mask & pixel to edit
		x0 = input[(row - 1) * numcols + (col - 1)];	// leftup
		x1 = input[(row + 1) * numcols + col];			// up
		x2 = input[(row - 1) * numcols + (col + 1)];	// rightup
		x3 = input[row * numcols + (col - 1)];			// left
		x5 = input[row * numcols + (col + 1)];			// right
		x6 = input[(row + 1) * numcols + (col - 1)];	// leftdown
		x7 = input[(row + -1) * numcols + col];			// down
		x8 = input[(row + 1) * numcols + (col + 1)];	// rightdown


		// calculate gradients for horizontal and vertical axis using sobel mask
		gradient_h = (x0 * -1) + (x2 * 1) + (x3 * -2) + (x5 * 2) + (x6 * -1) + (x8 * 1);
		gradient_v = (x0 * -1) + (x1 * -2) + (x3 * -1) + (x6 * 1) + (x7 * 2) + (x8 * 1);

		// using pythagoras theorem calculate the total gradient
		gradient = sqrt((gradient_h * gradient_h) + (gradient_v * gradient_v));

		if (gradient >= thresh)
		{
			gradient = 255;
		}
		else {
			gradient = 0;
		}
		// output pixel data
		output[row * numcols + col] = gradient;
	}
}

void load_image() {
	// read pgm file
	pgmread("image512x512.pgm", (void *)image, WIDTH, HEIGHT);
	// write read data for test
	pgmwrite("image_test512x512.pgm", (void *)image, WIDTH, HEIGHT);
}

void save_image() {
	// save pgm file
	pgmwrite("final512x512.pgm", (void *)final, WIDTH, HEIGHT);
}

void call_kernel() {
	int x, y;
	float *d_input, *d_output;

	printf("Block size: %dx%d\n", BLOCK_W, BLOCK_H);

	// memory size to allocate
	size_t memSize = WIDTH * HEIGHT * sizeof(float);

	// allocate memory
	hipMalloc(&d_input, memSize);
	hipMalloc(&d_output, memSize);

	// set value of pixels to 0
	for (y = 0; y < HEIGHT; y++) {
		for (x = 0; x < WIDTH; x++) {
			final[x][y] = 0.0;
		}
	}

	printf("Blocks per grid (width): %d\n", (WIDTH / BLOCK_W));
	printf("Blocks per grid (height): %d\n", (HEIGHT / BLOCK_H));

	// start time for blur
	start_time_blur = get_current_time();

	// copy image data into host input
	hipMemcpy(d_input, image, memSize, hipMemcpyHostToDevice);

	// call kernel blur
	dim3 threads(BLOCK_W, BLOCK_H); // threads per block
	dim3 blocks(WIDTH / BLOCK_W, HEIGHT / BLOCK_H); // blocks per grid 
	imageBlur << <blocks, threads >> > (d_input, d_output, WIDTH, HEIGHT);

	// sync threads
	hipDeviceSynchronize();

	// copy output back into input
	hipMemcpy(d_input, d_output, memSize, hipMemcpyDeviceToHost);

	// end time of blur filter
	end_time_blur = get_current_time();

	// start time for sobel filter
	start_time_sobel = get_current_time();

	// call sobel kernel with same threads and blocks amount
	sobelFilter << <blocks, threads >> > (d_input, d_output, WIDTH, HEIGHT);

	// sync threads
	hipDeviceSynchronize();

	// get output of kernel
	hipMemcpy(final, d_output, memSize, hipMemcpyDeviceToHost);

	// end time for sobel filter
	end_time_sobel = get_current_time();

	// if an error, display
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", "Main Loop", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// free memory
	hipFree(d_input);
	hipFree(d_output);
}

// main method to call methods
int main(int argc, char *argv[])
{
	// total time (parallel + serial)
	start_time_total = get_current_time();

	// firstly load image into buffer
	load_image();

	// invoke kernels
	call_kernel();

	// save output
	save_image();
	
	// end total time (parallel + serial)
	end_time_total = get_current_time();

	// print times
	double total = (end_time_sobel - start_time_sobel) + (end_time_blur - start_time_blur);

	printf("GPU blur (Including Data Transfer): %fs\n", end_time_blur - start_time_blur);
	printf("GPU sobel (Including Data Transfer): %fs\n", end_time_sobel - start_time_sobel);

	printf("Total P Time: %fs\n", total);
	printf("Total S Time: %fs\n", end_time_total - total);
	printf("Total S + P Time: %fs\n", end_time_total);

	// reset device
	hipDeviceReset();
	
	// exit with code 0
	return 0;
}